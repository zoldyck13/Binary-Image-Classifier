#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <filesystem>
#include <fstream>
#include <random>
using namespace cv;


const int IMG_SIZE = 32;
const int INPUT_SIZE = IMG_SIZE * IMG_SIZE;
const int HIDDEN_SIZE = 64;
const int NUM_CLASSES = 1;
const double LEARNING_RATE = 0.01;
const int EPOCHS = 500;


// ===================== CUDA DEVICE FUNCTIONS =====================
__device__ double sigmoid(double z){
    return 1.0 / (1.0 + exp(-z));
}

__device__ double sigmoid_derivative(double a){
    return a * (1 - a);
}

// ===================== CUDA KERNELS =====================
// Forward hidden layer
__global__ void forwardHiddenKernel(const double* d_input, double* d_hidden, const double* d_w_input_hidden, const double* d_b_hidden){
    int j = threadIdx.x;
    if(j < HIDDEN_SIZE){
        double sum = d_b_hidden[j];
        for(int i = 0; i < INPUT_SIZE; i++)
            sum += d_input[i] * d_w_input_hidden[i*HIDDEN_SIZE + j];
        d_hidden[j] = sigmoid(sum);
    }
}

// Forward output layer
__global__ void forwardOutputKernel(const double* d_hidden, double* d_output, const double* d_w_hidden_output, const double* d_b_output){
    int k = threadIdx.x;
    if(k < NUM_CLASSES){
        double sum = d_b_output[k];
        for(int j = 0; j < HIDDEN_SIZE; j++)
            sum += d_hidden[j] * d_w_hidden_output[j*NUM_CLASSES + k];
        d_output[k] = sigmoid(sum);
    }
}

// Compute delta output
__global__ void computeDeltaOutputKernel(const double* d_output, const double* d_target, double* d_delta_output){
    int k = threadIdx.x;
    if(k < NUM_CLASSES)
        d_delta_output[k] = (d_output[k] - d_target[k]) * sigmoid_derivative(d_output[k]);
}

// Compute delta hidden
__global__ void computeDeltaHiddenKernel(const double* d_w_hidden_output, const double* d_delta_output, const double* d_hidden, double* d_delta_hidden){
    int j = threadIdx.x;
    if(j < HIDDEN_SIZE){
        double sum = 0.0;
        for(int k = 0; k < NUM_CLASSES; k++)
            sum += d_delta_output[k] * d_w_hidden_output[j*NUM_CLASSES + k];
        d_delta_hidden[j] = sum * sigmoid_derivative(d_hidden[j]);
    }
}

// Update weights input->hidden
__global__ void updateWeightsInputHidden(double* d_w_input_hidden, const double* d_input, const double* d_delta_hidden){
    int i = threadIdx.x;
    int j = threadIdx.y;
    if(i < INPUT_SIZE && j < HIDDEN_SIZE)
        d_w_input_hidden[i*HIDDEN_SIZE + j] -= LEARNING_RATE * d_delta_hidden[j] * d_input[i];
}

// Update weights hidden->output
__global__ void updateWeightsHiddenOutput(double* d_w_hidden_output, const double* d_hidden, const double* d_delta_output){
    int j = threadIdx.x;
    int k = threadIdx.y;
    if(j < HIDDEN_SIZE && k < NUM_CLASSES)
        d_w_hidden_output[j*NUM_CLASSES + k] -= LEARNING_RATE * d_delta_output[k] * d_hidden[j];
}

// Update biases
__global__ void updateBiases(double* d_b_hidden, const double* d_delta_hidden,
                             double* d_b_output, const double* d_delta_output){
    int idx = threadIdx.x;
    if(idx < HIDDEN_SIZE) d_b_hidden[idx] -= LEARNING_RATE * d_delta_hidden[idx];
    if(idx < NUM_CLASSES) d_b_output[idx] -= LEARNING_RATE * d_delta_output[idx];
}



// ===================== DATASET LOADING =====================
void loadDataset(const std::string &path, std::vector<std::vector<double>> &X, std::vector<std::vector<double>> &Y){
    
    std::vector<std::string> categories = {"class_a", "class_b"}; 

    for(size_t class_index = 0; class_index < categories.size(); class_index++){
        std::string folder = path + "/" + categories[class_index];

        if(!std::filesystem::exists(folder)) {
            std::cerr << "Warning: folder \"" << folder << "\" does not exist!" << std::endl;
            continue;
        }

        int count = 0; 

        for(const auto &img_file : std::filesystem::directory_iterator(folder)){
            cv::Mat img = cv::imread(img_file.path().string());
            if(img.empty()) continue;

            cv::Mat gray, resized;
            cv::cvtColor(img, gray, cv::COLOR_BGR2GRAY);
            cv::resize(gray, resized, cv::Size(IMG_SIZE, IMG_SIZE));

            std::vector<double> input(INPUT_SIZE);
            for(int r = 0; r < IMG_SIZE; r++)
                for(int c = 0; c < IMG_SIZE; c++)
                    input[r*IMG_SIZE + c] = resized.at<uchar>(r,c) / 255.0;

            X.push_back(input);

            std::vector<double> output(NUM_CLASSES, class_index == 0 ? 1.0 : 0.0);
            Y.push_back(output);

            count++; 
        }

        if(count > 0) {
            std::cout << "Folder \"" << folder << "\" loaded with class_index " 
                      << class_index << " (" << count << " images)" << std::endl;
            std::cout.flush(); 
        }
    }
}


void saveNN(const std::string &filename,
            const std::vector<double> &w_input_hidden,
            const std::vector<double> &b_hidden,
            const std::vector<double> &w_hidden_output,
            const std::vector<double> &b_output) 
{
    std::ofstream myfile(filename);

    for(size_t i = 0; i < w_input_hidden.size(); i++)
        myfile << w_input_hidden[i] << "\n";

    for(size_t i = 0; i < b_hidden.size(); i++)
        myfile << b_hidden[i] << "\n";

    for(size_t i = 0; i < w_hidden_output.size(); i++)
        myfile << w_hidden_output[i] << "\n";

    for(size_t i = 0; i < b_output.size(); i++)
        myfile << b_output[i] << "\n";

    std::cout << "Network saved to " << filename << std::endl;
}



int main(){
    std::vector<std::vector<double>> X, Y;
    loadDataset("dataset", X, Y);
    std::cout << "Loaded " << X.size() << " samples." << std::endl;

    // ===================== INITIALIZE NETWORK =====================
    std::vector<double> w_input_hidden(INPUT_SIZE*HIDDEN_SIZE);
    std::vector<double> b_hidden(HIDDEN_SIZE,0.0);
    std::vector<double> w_hidden_output(HIDDEN_SIZE*NUM_CLASSES);
    std::vector<double> b_output(NUM_CLASSES,0.0);

	static std::random_device rd;
	static std::mt19937 gen(rd());
	double limit = std::sqrt(6.0 / (INPUT_SIZE+HIDDEN_SIZE));
	double limit1 = std::sqrt(6.0 / (HIDDEN_SIZE+NUM_CLASSES));
	double limit2 = std::sqrt(6.0 / (HIDDEN_SIZE));
	double limit3 = std::sqrt(6.0 / (NUM_CLASSES));
	std::uniform_real_distribution<double> dist(-limit, limit);
	std::uniform_real_distribution<double> dist1(-limit1, limit1);
	std::uniform_real_distribution<double> dist2(-limit2, limit2);
	std::uniform_real_distribution<double> dist3(-limit3, limit3);


    for(auto &w : w_input_hidden) w = dist(gen);
    for(auto &w : w_hidden_output) w = dist1(gen);
	for(auto &b : b_hidden) b = dist2(gen);
	for(auto &b : b_output) b = dist3(gen);

    // ===================== ALLOCATE GPU MEMORY =====================
    double *d_input, *d_hidden, *d_output;
    double *d_w_input_hidden, *d_b_hidden, *d_w_hidden_output, *d_b_output;
    double *d_delta_hidden, *d_delta_output;
    double *d_target;

    hipMalloc(&d_input, INPUT_SIZE*sizeof(double));
    hipMalloc(&d_hidden, HIDDEN_SIZE*sizeof(double));
    hipMalloc(&d_output, NUM_CLASSES*sizeof(double));
    hipMalloc(&d_w_input_hidden, INPUT_SIZE*HIDDEN_SIZE*sizeof(double));
    hipMalloc(&d_b_hidden, HIDDEN_SIZE*sizeof(double));
    hipMalloc(&d_w_hidden_output, HIDDEN_SIZE*NUM_CLASSES*sizeof(double));
    hipMalloc(&d_b_output, NUM_CLASSES*sizeof(double));
    hipMalloc(&d_delta_hidden, HIDDEN_SIZE*sizeof(double));
    hipMalloc(&d_delta_output, NUM_CLASSES*sizeof(double));
    hipMalloc(&d_target, NUM_CLASSES*sizeof(double));

    hipMemcpy(d_w_input_hidden, w_input_hidden.data(), INPUT_SIZE*HIDDEN_SIZE*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b_hidden, b_hidden.data(), HIDDEN_SIZE*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_w_hidden_output, w_hidden_output.data(), HIDDEN_SIZE*NUM_CLASSES*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b_output, b_output.data(), NUM_CLASSES*sizeof(double), hipMemcpyHostToDevice);

    // ===================== TRAINING =====================
    for(int epoch=0; epoch<EPOCHS; epoch++){
        double total_loss = 0.0;

        for(size_t sample=0; sample<X.size(); sample++){
            
            hipMemcpy(d_input, X[sample].data(), INPUT_SIZE*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_target, Y[sample].data(), NUM_CLASSES*sizeof(double), hipMemcpyHostToDevice);

            // Forward pass
            forwardHiddenKernel<<<1,HIDDEN_SIZE>>>(d_input,d_hidden,d_w_input_hidden,d_b_hidden);
            hipDeviceSynchronize();
            forwardOutputKernel<<<1,NUM_CLASSES>>>(d_hidden,d_output,d_w_hidden_output,d_b_output);
            hipDeviceSynchronize();

            
            computeDeltaOutputKernel<<<1,NUM_CLASSES>>>(d_output,d_target,d_delta_output);
            hipDeviceSynchronize();
            computeDeltaHiddenKernel<<<1,HIDDEN_SIZE>>>(d_w_hidden_output,d_delta_output,d_hidden,d_delta_hidden);
            hipDeviceSynchronize();

           
            dim3 threadsIH(INPUT_SIZE,HIDDEN_SIZE);
            updateWeightsInputHidden<<<1,threadsIH>>>(d_w_input_hidden,d_input,d_delta_hidden);
            hipDeviceSynchronize();

            dim3 threadsHO(HIDDEN_SIZE,NUM_CLASSES);
            updateWeightsHiddenOutput<<<1,threadsHO>>>(d_w_hidden_output,d_hidden,d_delta_output);
            hipDeviceSynchronize();

            updateBiases<<<1,HIDDEN_SIZE>>>(d_b_hidden,d_delta_hidden,d_b_output,d_delta_output);
            hipDeviceSynchronize();
        }

        if(epoch%50==0) std::cout<<"Epoch "<<epoch<<" done."<<std::endl;
    }

    std::cout<<"Training complete."<<std::endl;

    // ===================== COPY WEIGHTS BACK TO HOST =====================
    hipMemcpy(w_input_hidden.data(),d_w_input_hidden,INPUT_SIZE*HIDDEN_SIZE*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(b_hidden.data(),d_b_hidden,HIDDEN_SIZE*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(w_hidden_output.data(),d_w_hidden_output,HIDDEN_SIZE*NUM_CLASSES*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(b_output.data(),d_b_output,NUM_CLASSES*sizeof(double),hipMemcpyDeviceToHost);
	

    // ===================== SAVE THE NETWORK ================
    saveNN("Network.csv", w_input_hidden, b_hidden, w_hidden_output, b_output);


    // ===================== CLEANUP =====================
    hipFree(d_input); hipFree(d_hidden); hipFree(d_output);
    hipFree(d_w_input_hidden); hipFree(d_b_hidden);
    hipFree(d_w_hidden_output); hipFree(d_b_output);
    hipFree(d_delta_hidden); hipFree(d_delta_output); hipFree(d_target);

    return 0;
}
